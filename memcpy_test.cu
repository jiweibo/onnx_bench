#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

#include <future>
#include <iostream>
#include <thread>

#include "thread_pool.h"
#include "utils/barrier.h"
#include "utils/util.h"

#include <gflags/gflags.h>
#include <glog/logging.h>

DEFINE_int32(thread_num, 4, "thread number");
DEFINE_int32(size, 32 * 1024 * 1024, "memory size");
DEFINE_int32(warmup, 0, "warmup");
DEFINE_int32(repeats, 1, "repeats");
DEFINE_bool(pin, false, "use pin memory, default is false");
DEFINE_string(modes, "h2d d2h d2d run", "h2d d2h d2d run");

namespace {

void Copy(void* dst, const void* src, size_t size, hipMemcpyKind kind,
          hipStream_t stream, Barrier* barrier, int repeats = 1) {
  for (size_t i = 0; i < repeats; ++i) {
    if (barrier) {
      barrier->Wait();
    }
    CUDA_CHECK(hipMemcpyAsync(dst, src, size, kind, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));
  }
}

__global__ void func(float* device_data, size_t size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    device_data[idx] = device_data[idx] * 0.9f + 0.1f;
  }
}

void RunKernel(float* device_data, size_t size, hipStream_t stream,
               Barrier* barrier, int repetas = 1) {
  dim3 threads_per_block = 512;
  dim3 blocks_per_grid = (size + threads_per_block.x - 1) / threads_per_block.x;
  for (size_t i = 0; i < repetas; ++i) {
    if (barrier) {
      barrier->Wait();
    }
    func<<<blocks_per_grid, threads_per_block, 0, stream>>>(device_data, size);
    CUDA_CHECK(hipStreamSynchronize(stream));
  }
}

} // namespace

int main(int argc, char** argv) {
  gflags::ParseCommandLineFlags(&argc, &argv, true);

  Barrier barrier(FLAGS_thread_num);
  ThreadPool tp(FLAGS_thread_num);
  std::vector<hipStream_t> streams(FLAGS_thread_num);
  std::vector<float*> host_datas(FLAGS_thread_num);
  std::vector<float*> device_datas(FLAGS_thread_num);
  std::vector<float*> device_datas2(FLAGS_thread_num);

  for (size_t i = 0; i < streams.size(); ++i) {
    CUDA_CHECK(hipStreamCreate(&streams[i]));
    if (FLAGS_pin) {
      CUDA_CHECK(hipHostMalloc(&host_datas[i], FLAGS_size));
    } else {
      host_datas[i] = (float*)malloc(FLAGS_size);
    }
    memset(host_datas[i], 0, FLAGS_size);
    CUDA_CHECK(hipMalloc((void**)&device_datas[i], FLAGS_size));
    CUDA_CHECK(hipMalloc((void**)&device_datas2[i], FLAGS_size));
  }

  auto modes = SplitToStringVec(FLAGS_modes, ' ');
  if (modes.empty() || modes.size() == 1) {
    bool empty = modes.empty();
    modes.resize(FLAGS_thread_num);
    for (size_t i = 1; i < FLAGS_thread_num; ++i) {
      modes[i] = empty ? "h2d" : modes[0];
    }
  } else {
    CHECK_EQ(modes.size(), FLAGS_thread_num);
  }

  // warmup
  tp.enqueue([&]() {
      Copy(device_datas[0], host_datas[0], FLAGS_size, hipMemcpyHostToDevice,
           streams[0], nullptr, FLAGS_warmup);
      Copy(host_datas[0], device_datas[0], FLAGS_size, hipMemcpyDeviceToHost,
           streams[0], nullptr, FLAGS_warmup);
      Copy(device_datas2[0], device_datas[0], FLAGS_size,
           hipMemcpyDeviceToDevice, streams[0], nullptr, FLAGS_warmup);
      RunKernel(device_datas[0], FLAGS_size / sizeof(float), streams[0],
                nullptr, FLAGS_warmup);
    }).get();
  LOG(INFO) << "Warmup done.";

  std::vector<std::future<void>> rets(FLAGS_thread_num);
  for (size_t i = 0; i < FLAGS_thread_num; ++i) {
    rets[i] = tp.enqueue(
        [&](int idx) {
          void *dst{nullptr}, *src{nullptr};
          hipMemcpyKind kind;
          if (modes[idx] == "h2d") {
            dst = device_datas[idx];
            src = host_datas[idx];
            kind = hipMemcpyHostToDevice;
          } else if (modes[idx] == "d2h") {
            dst = host_datas[idx];
            src = device_datas[idx];
            kind = hipMemcpyDeviceToHost;
          } else if (modes[idx] == "d2d") {
            dst = device_datas[idx];
            src = device_datas[idx];
            kind = hipMemcpyDeviceToDevice;
          } else if (modes[idx] == "run") {
            RunKernel(device_datas[idx], FLAGS_size / sizeof(float),
                      streams[idx], &barrier, FLAGS_repeats);
            return;
          } else {
            LOG(FATAL) << "Not supported mode: " << modes[idx];
          }
          Copy(dst, src, FLAGS_size, kind, streams[idx], &barrier,
               FLAGS_repeats);
        },
        i);
  }
  for (size_t i = 0; i < FLAGS_thread_num; ++i) {
    rets[i].get();
  }
  LOG(INFO) << "Run done.";

  for (size_t i = 0; i < FLAGS_thread_num; ++i) {
    CUDA_CHECK(hipStreamDestroy(streams[i]));
    CUDA_CHECK(hipFree(device_datas[i]));
    CUDA_CHECK(hipFree(device_datas2[i]));
    if (FLAGS_pin) {
      CUDA_CHECK(hipHostFree(host_datas[i]));
    } else {
      free(host_datas[i]);
    }
  }
  return 0;
}